
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void suma(int *a, int *b, int *c){

*c = *a + *b;
}

int main(){

int a, b, c;
int *gpu_a, *gpu_b, *gpu_c;

printf("Indica el valor de a: ");
scanf("%d", &a);
printf("Indica el valor de b: ");
scanf("%d", &b);


hipMalloc((void**)&gpu_a, sizeof(int));
hipMalloc((void**)&gpu_b, sizeof(int));
hipMalloc((void**)&gpu_c, sizeof(int));

hipMemcpy(gpu_a, &a, sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(gpu_b, &b, sizeof(int), hipMemcpyHostToDevice);

suma<<<1,1>>>(gpu_a, gpu_b, gpu_c);
hipMemcpy(&c, gpu_c, sizeof(int), hipMemcpyDeviceToHost);
printf("%d + %d = %d \n", a, b, c);

hipFree(gpu_a);
hipFree(gpu_b);
hipFree(gpu_c);

return 0;
}
