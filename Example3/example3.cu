
#include <hip/hip_runtime.h>
#include <stdio.h>

#define  N  1000

struct Matrix{
int fil;
int col;
double* data;
};


__global__ void sqr_matrix_mult(double *matrix_a, double *matrix_b, double *matrix_result, int size){

int i, j, iter, index;

for (i = blockIdx.x; i < size; i = i + gridDim.x){
	for (j = blockIdx.y; j < size; j = j + gridDim.y){
		
		index = i*size+j;
		matrix_result[index] = 0.0;

		for (iter = 0; iter < size; iter++ ) {

		matrix_result[index] = matrix_result[index]
		+ matrix_a[size*i+iter] * matrix_b[size*iter+j];

		}
	}
}
}

void matrix_print(Matrix mat);


int main(void){

Matrix A, B, C;
double *gpu_A, *gpu_B, *gpu_C;

dim3 grid(1000,1000);

A.fil = N; A.col = N;
A.data =(double*) malloc(sizeof(double)*N*N);

B.fil = N; B.col = N;
B.data = (double*) malloc(sizeof(double)*N*N);

C.fil = N; C.col = N;
C.data = (double*) malloc(sizeof(double)*N*N);

//Reservamos memoria en la GPU
hipMalloc((void**) &gpu_A, N*N*sizeof(double));
hipMalloc((void**) &gpu_B, N*N*sizeof(double));
hipMalloc((void**) &gpu_C, N*N*sizeof(double));

//Rellenamos las matrices A y B
for (int i=0; i<N; i++){
	for (int j=0; j<N; j++){
		A.data[i*N+j]=i+j;
		B.data[i*N+j]=i-j;
}
}

//Enviamos nuestras matrices a la GPU
hipMemcpy(gpu_A, A.data, N*N*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(gpu_B, B.data, N*N*sizeof(double), hipMemcpyHostToDevice);

//Lanzamos nuestra función
sqr_matrix_mult<<<grid,1>>>(gpu_A,gpu_B,gpu_C,N);

//Extraemos nuestro resultado de la GPU
hipMemcpy(C.data, gpu_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

//Imprimimos el resultado con una función auxiliar
printf("A=\n");
matrix_print(A);
printf("\nB=\n");
matrix_print(B);
printf("\nA*B=\n");
matrix_print(C);

//Liberamos la memoria
hipFree(gpu_A);
hipFree(gpu_B);
hipFree(gpu_C);
free(A.data);
free(B.data);
free(C.data);
return 0;
}

void matrix_print(Matrix mat){
for (int i = 0; i < mat.fil; i++){
	for(int j = 0; j < mat.col; j++){
		if (mat.data[i*N+j] < 0){
			printf("%lf ", mat.data[i*N+j]);
		}
		else{
			printf(" %lf ", mat.data[i*N+j]);
		}
	
		
}
	printf("\n");
}

}
