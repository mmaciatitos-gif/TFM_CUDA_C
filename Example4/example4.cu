
#include <hip/hip_runtime.h>
#include <stdio.h>

#define  N  1000

struct Matrix{
int fil;
int col;
double* data;
};


__global__ void sqr_matrix_mult(double *matrix_a, double *matrix_b, double *matrix_result, int size){

int i, j, iter, index;

for (i = blockIdx.x; i < size; i = i + gridDim.x){
	for (j = blockIdx.y; j < size; j = j + gridDim.y){
		
		index = i*size+j;
		matrix_result[index] = 0.0;

		for (iter = 0; iter < size; iter++ ) {

		matrix_result[index] = matrix_result[index]
		+ matrix_a[size*i+iter] * matrix_b[size*iter+j];

		}
	}
}
}


int main(void){

Matrix A, B, C;
double *gpu_A, *gpu_B, *gpu_C;

dim3 grid(1000,1000);

A.fil = N; A.col = N;
A.data =(double*) malloc(sizeof(double)*N*N);

B.fil = N; B.col = N;
B.data = (double*) malloc(sizeof(double)*N*N);

C.fil = N; C.col = N;
C.data = (double*) malloc(sizeof(double)*N*N);

//Reservamos memoria en la GPU
hipMalloc((void**) &gpu_A, N*N*sizeof(double));
hipMalloc((void**) &gpu_B, N*N*sizeof(double));
hipMalloc((void**) &gpu_C, N*N*sizeof(double));

//Rellenamos las matrices A y B
for (int i=0; i<N; i++){
	for (int j=0; j<N; j++){
		A.data[i*N+j]=i+j;
		B.data[i*N+j]=i-j;
}
}

//Creamos las variables evento
hipEvent_t  start, stop;

//Creamos los eventos
hipEventCreate (&start);
hipEventCreate (&stop);

//Creamos una variable para almacenar los tiempos y otra para hacer media:
float timeTemp = 0;
float timeAvg = 0;

//Creamos un bucle
for(int i=0; i<1000; i++){

//Grabamos el evento inicial
hipEventRecord(start,0);

//Enviamos nuestras matrices a la GPU
hipMemcpy(gpu_A, A.data, N*N*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(gpu_B, B.data, N*N*sizeof(double), hipMemcpyHostToDevice);

//Lanzamos nuestra función
sqr_matrix_mult<<<grid,1>>>(gpu_A,gpu_B,gpu_C,N);

//Extraemos nuestro resultado de la GPU
hipMemcpy(C.data, gpu_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

//Grabamos el evento final
hipEventRecord(stop,0);
hipEventSynchronize(stop);

//Obtenemos el tiempo de ejecucion
hipEventElapsedTime(&timeTemp, start, stop);
timeAvg = timeAvg + timeTemp;
}

//Destruimos los eventos
hipEventDestroy(start);
hipEventDestroy(stop);

//Imprimimos la media de tiempo de ejecucion
printf("Tiempo: %f\n", timeAvg/1000.0);

//Liberamos la memoria
hipFree(gpu_A);
hipFree(gpu_B);
hipFree(gpu_C);
free(A.data);
free(B.data);
free(C.data);
return 0;
}
